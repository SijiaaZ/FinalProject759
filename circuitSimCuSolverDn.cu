// Apapted from https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuSOLVER/Xpotrf/cusolver_Xpotrf_example.cu

#include <cstdio>
#include <cstdlib>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include "cusolver_utils.h"

#include "parse.h"
#include "matrix_helper.h"


int main(int argc, char *argv[]) {
    char* filename=argv[1];

    int elementListLength=0;
    Element* elementList=parseNetlist(filename, elementListLength);
    printf("Success:%d\n",elementList==NULL);
    printf("element List Length:%d\n",elementListLength);
    for(int i=0;i<elementListLength;i++)
    {
        printf("Node1:%d,Node2:%d,value:%.3f\n",elementList[i].Node1,elementList[i].Node2,elementList[i].value);
    }

    int augmented_matrix_dim=get_Matrix_Dim_from_nodes(elementList,elementListLength);
    printf("%d\n",augmented_matrix_dim);
    std::vector<double> conductance(augmented_matrix_dim*augmented_matrix_dim);
    std::vector<double> currents(augmented_matrix_dim);
    elementList_to_augmented_Matrix(elementList, elementListLength, conductance, currents, augmented_matrix_dim);
    for(int i=0;i<augmented_matrix_dim;i++)
    {
        for(int j=0;j<augmented_matrix_dim;j++)
        {
            printf("%f,",conductance[i*augmented_matrix_dim+j]);
        }
        printf("\n");
    }
    for(int i=0;i<augmented_matrix_dim;i++)
    {
        printf("%f\n",currents[i]);
    }
    int matrix_dim=augmented_matrix_dim-1;
    std::vector<double> conductance_definite((augmented_matrix_dim-1)*(augmented_matrix_dim-1));
    std::vector<double> currents_definite(augmented_matrix_dim-1);
    augmented_Matrix_to_definite_matrix( elementListLength,  conductance,  currents,  conductance_definite, currents_definite,  augmented_matrix_dim);
    for(int i=0;i<matrix_dim;i++)
    {
        for(int j=0;j<matrix_dim;j++)
        {
            printf("%f,",conductance_definite[i*matrix_dim+j]);
        }
        printf("\n");
    }
    std::vector<double> A=conductance_definite;
    std::vector<double> B=currents_definite;

    hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;

    using data_type = double;

    const int64_t m = matrix_dim;
    const int64_t lda = m;
    const int64_t ldb = m;

    /*
     *     | 1     2     3 |
     * A = | 2     5     5 | = L0 * L0**T
     *     | 3     5    12 |
     *
     *            | 1.0000         0         0 |
     * where L0 = | 2.0000    1.0000         0 |
     *            | 3.0000   -1.0000    1.4142 |
     *
     */

    //const std::vector<data_type> A = {1.0, 2.0, 3.0, 2.0, 5.0, 5.0, 3.0, 5.0, 12.0};
    //const std::vector<data_type> B = {1.0, 2.0, 3.0};
    std::vector<data_type> X(m, 0);
    std::vector<data_type> L(lda * m, 0);
    int info = 0;

    data_type *d_A = nullptr; /* device copy of A */
    data_type *d_B = nullptr; /* device copy of B */
    int *d_info = nullptr;    /* error info */

    size_t d_lwork = 0;     /* size of workspace */
    void *d_work = nullptr; /* device workspace */
    size_t h_lwork = 0;     /* size of workspace */
    void *h_work = nullptr; /* host workspace */

    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    std::printf("A = (matlab base-1)\n");
    print_matrix(m, m, A.data(), lda);
    std::printf("=====\n");

    std::printf("B = (matlab base-1)\n");
    print_matrix(m, 1, B.data(), ldb);
    std::printf("=====\n");

    /* step 1: create cusolver handle, bind a stream */
    CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

    /* step 2: copy A to device */
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(data_type) * A.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_B), sizeof(data_type) * B.size()));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int)));

    CUDA_CHECK(hipMemcpyAsync(d_A, A.data(), sizeof(data_type) * A.size(), hipMemcpyHostToDevice,
                               stream));
    CUDA_CHECK(hipMemcpyAsync(d_B, B.data(), sizeof(data_type) * B.size(), hipMemcpyHostToDevice,
                               stream));

    /* step 3: query working space */
    CUSOLVER_CHECK(hipsolverDnXpotrf_bufferSize(
        cusolverH, NULL, uplo, m, traits<data_type>::cuda_data_type, d_A, lda,
        traits<data_type>::cuda_data_type, &d_lwork, &h_lwork));

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(data_type) * d_lwork));

    /* step 4: Cholesky factorization */
    CUSOLVER_CHECK(hipsolverDnXpotrf(cusolverH, NULL, uplo, m, traits<data_type>::cuda_data_type,
                                    d_A, lda, traits<data_type>::cuda_data_type, d_work, d_lwork,
                                    h_work, h_lwork, d_info));

    CUDA_CHECK(hipMemcpyAsync(L.data(), d_A, sizeof(data_type) * A.size(), hipMemcpyDeviceToHost,
                               stream));
    CUDA_CHECK(hipMemcpyAsync(&info, d_info, sizeof(int), hipMemcpyDeviceToHost, stream));

    CUDA_CHECK(hipStreamSynchronize(stream));

    std::printf("after Xpotrf: info = %d\n", info);
    if (0 > info) {
        std::printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }

    std::printf("L = (matlab base-1)\n");
    print_matrix(m, m, L.data(), lda);
    std::printf("=====\n");

    /*
     * step 5: solve A*X = B
     *       | 1 |       | -0.3333 |
     *   B = | 2 |,  X = |  0.6667 |
     *       | 3 |       |  0      |
     *
     */

    CUSOLVER_CHECK(hipsolverDnXpotrs(cusolverH, NULL, uplo, m, 1, /* nrhs */
                                    traits<data_type>::cuda_data_type, d_A, lda,
                                    traits<data_type>::cuda_data_type, d_B, ldb, d_info));

    CUDA_CHECK(hipMemcpyAsync(X.data(), d_B, sizeof(data_type) * X.size(), hipMemcpyDeviceToHost,
                               stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    std::printf("X = (matlab base-1)\n");
    print_matrix(m, 1, X.data(), ldb);
    std::printf("=====\n");

    /* free resources */
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_info));
    CUDA_CHECK(hipFree(d_work));

    CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

    CUDA_CHECK(hipStreamDestroy(stream));

    CUDA_CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}