#include "gmres.h"
#include <iostream>

int main(int argc, char *argv[]) {

    hipStream_t stream1;
    hipStreamCreate(&stream1);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    int matrix_dim=4;

    double* A_h=new double[matrix_dim * matrix_dim]{1,-0.5,0,0,0,1,0,0,0,0,1,0,0.5,0,0,1};
    double* A;
    hipMallocManaged(&A, sizeof(double) * matrix_dim * matrix_dim);
    hipMemcpy(A, A_h, sizeof(double) * matrix_dim * matrix_dim, hipMemcpyDefault);

    double* b_h=new double[matrix_dim]{0.35,0.25,0.2,0.1};
    double* b;
    hipMallocManaged(&b, sizeof(double) * matrix_dim );
    hipMemcpy(b, b_h, sizeof(double) * matrix_dim , hipMemcpyDefault);

    double* Q;
    hipMallocManaged(&Q, sizeof(double) * matrix_dim * matrix_dim);
    for(int i=0;i<matrix_dim*matrix_dim;i++)
    {
        Q[i]=0;
    }

    double* H;
    hipMallocManaged(&H, sizeof(double) * matrix_dim * matrix_dim);
    for(int i=0;i<matrix_dim*matrix_dim;i++)
    {
        H[i]=0;
    }

    double *x;
    hipMallocManaged(&x, sizeof(double) * matrix_dim );


     GMRES(handle,stream1,A, b, x, Q,  H,matrix_dim,4, 0.001);


    // printf("A===================\n");
    // for(int i=0;i<matrix_dim;i++)
    // {
    //     for(int j=0;j<matrix_dim;j++)
    //     {
    //         printf("%.3f,",A[IDX2C(i,j,matrix_dim)]);
    //     }
    //     printf("\n");
    // }
    // printf("Q===================\n");

    // for(int i=0;i<matrix_dim;i++)
    // {
    //     for(int j=0;j<matrix_dim;j++)
    //     {
    //         printf("%.3f,",Q[IDX2C(i,j,matrix_dim)]);
    //     }
    //     printf("\n");
    // }

    // printf("H===================\n");
    // for(int i=0;i<matrix_dim;i++)
    // {
    //     for(int j=0;j<matrix_dim;j++)
    //     {
    //         printf("%.3f,",H[IDX2C(i,j,matrix_dim)]);
    //     }
    //     printf("\n");
    // }


    
    
    hipFree(A);
    hipFree(Q);
    hipFree(H);
    hipblasDestroy(handle);
    hipStreamDestroy(stream1);

    delete []A_h;
    delete []b_h;

    return 0;
}