#include "gmres.h"
#include <iostream>

int main(int argc, char *argv[]) {

    hipStream_t stream1;
    hipStreamCreate(&stream1);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    int matrix_dim=4;

    double* A_h=new double[matrix_dim * matrix_dim]{1,-0.5,0,0,0,1,0,0,0,0,1,0,0.5,0,0,1};
    double* A;
    hipMallocManaged(&A, sizeof(double) * matrix_dim * matrix_dim);
    hipMemcpy(A, A_h, sizeof(double) * matrix_dim * matrix_dim, hipMemcpyDefault);

    double* b_h=new double[matrix_dim]{0.35,0.25,0.2,0.1};
    double* b;
    hipMallocManaged(&b, sizeof(double) * matrix_dim );
    hipMemcpy(b, b_h, sizeof(double) * matrix_dim , hipMemcpyDefault);

    double* Q;
    hipMallocManaged(&Q, sizeof(double) * matrix_dim * matrix_dim);
    for(int i=0;i<matrix_dim*matrix_dim;i++)
    {
        Q[i]=0;
    }
    std::cout<<Q<<std::endl;
    std::cout<<(double*) (Q+matrix_dim)<<std::endl;
    std::cout<<(double*) (Q+matrix_dim)-Q<<std::endl;
    std::cout<<&Q[matrix_dim]<<std::endl;
    std::cout<<&Q[matrix_dim]-Q<<std::endl;

    double* H;
    hipMallocManaged(&H, sizeof(double) * matrix_dim * matrix_dim);
    for(int i=0;i<matrix_dim*matrix_dim;i++)
    {
        H[i]=0;
    }

    double *x;
    hipMallocManaged(&x, sizeof(double) * matrix_dim );


    GMRES(handle,A, b, x, Q, H, matrix_dim,stream1);

    for(int i=0;i<matrix_dim;i++)
    {
        for(int j=0;j<matrix_dim;j++)
        {
            printf("%.3f,",A[IDX2C(i,j,matrix_dim)]);
        }
        printf("\n");
    }
    printf("===================\n");

    for(int i=0;i<matrix_dim;i++)
    {
        for(int j=0;j<matrix_dim;j++)
        {
            printf("%.3f,",Q[IDX2C(i,j,matrix_dim)]);
        }
        printf("\n");
    }


    
    
    hipFree(A);
    hipFree(Q);
    hipFree(H);
    hipblasDestroy(handle);
    hipStreamDestroy(stream1);

    delete []A_h;
    delete []b_h;

    return 0;
}