#include "hip/hip_runtime.h"
#include "gmres.h"
#include <hip/hip_runtime.h>
void cublasCheck(hipblasStatus_t stat, const char* function_name)
{
    //printf("%s\n",function_name);
    if(stat!=HIPBLAS_STATUS_SUCCESS)
        printf("%s failed\n",function_name);
}



__global__ void element_append_vector(double* h, int k, double value)
{
    //printf("element_append_vector\n");
    h[k+1]=value;
}

void GMRES(hipblasHandle_t handle,const double* A, double*b, double* x, double* Q, double* H,const int matrix_dim,hipStream_t stream_id)
{
    hipblasStatus_t cudaStat;
    cudaStat=hipblasSetStream(handle, stream_id);

    double alpha=-1;
    double beta=1;

    double *r;
    hipMallocManaged(&r, sizeof(double) * matrix_dim);
    hipMemcpy(r,b,sizeof(double) *matrix_dim,hipMemcpyDefault);

    //r=b-A*x;
    cudaStat=hipblasDgemv(handle, HIPBLAS_OP_N,
                           matrix_dim, matrix_dim,
                           &alpha,
                           A, matrix_dim,
                           x, 1,
                           &beta,
                           r, 1);

    cublasCheck(cudaStat,"hipblasDgemv");

    //r_norm = norm(r);
    double r_norm=0;//r_norm is on the host memory
    cudaStat = hipblasDnrm2( handle, matrix_dim,
                            r, 1, &r_norm);//probably blocking to make sure the correct r_norm is on the host
    cublasCheck(cudaStat,"hipblasDnrm2");

    double r_norm_reciprocal=0;//r_norm_reciprocal is on the host memory
    if(r_norm!=0)
    {
        r_norm_reciprocal=1/r_norm;//do on the host
    }

    //r = r / r_norm;
    cudaStat = hipblasDscal(handle, matrix_dim,
                            &r_norm_reciprocal,
                            r, 1);
    cublasCheck(cudaStat,"hipblasDscal");

    hipMemcpy(Q,r,sizeof(double) *matrix_dim,hipMemcpyDefault);

    int k=0;
    arnoldi(handle, A,  Q, H, k, matrix_dim,stream_id);

    hipFree(r);
}

// A (device) is stored in column-major order, Q (device) is 2D array, Q[i] means Qth column
// k is the total finished column index
// q vector (device) has dimension matrix_dim; h vector (device) has dimension matrix_dim+1
void arnoldi(hipblasHandle_t handle,const double* A, double* Q, double *H, const int k, const int matrix_dim,hipStream_t stream_id)
{
    hipblasStatus_t cudaStat;
    cudaStat=hipblasSetStream(handle, stream_id);
    cublasCheck(cudaStat,"hipblasSetStream");


    double alpha=1;
    double beta=0;

    double*q;
    hipMallocManaged(&q, sizeof(double) * matrix_dim);
    for(int i=0;i<matrix_dim;i++)
    {
        q[i]=0;
    }
    double* h;
    hipMallocManaged(&h, sizeof(double) * matrix_dim);
    for(int i=0;i<matrix_dim;i++)
    {
        h[i]=0;
    }


    double* q_norm;
    hipMallocManaged(&q_norm, sizeof(double) * 1);
    
    
    // debugging printf
    // for(int i=0;i<matrix_dim;i++)
    // {
    //     for(int j=0;j<matrix_dim;j++)
    //     {
    //         printf("%.3f,",Q[IDX2C(i,j,matrix_dim)]);
    //     }
    //     printf("\n");
    // }

    //q = A*Q(:,k);
    cudaStat=hipblasDgemv(handle, HIPBLAS_OP_N,
                           matrix_dim, matrix_dim,
                           &alpha,
                           A, matrix_dim,
                           (double*) (Q+k*matrix_dim), 1,
                           &beta,
                           q, 1);
    cublasCheck(cudaStat,"hipblasDgemv");   

    for(int i=0;i<k+1;i++)
    {
        // h(i) = q' * Q(:, i);
        cudaStat=hipblasDdot (handle, matrix_dim,
                           q, 1,
                           (double*)(Q+i*matrix_dim), 1,
                           (double*)(h+i));
                        
        cublasCheck(cudaStat,"hipblasDdot");

        alpha=-*(h+i);//alpha should be the const in hipblasDaxpy so the hipDeviceSynchronize must be added
        //q = q - h(i) * Q(:, i);
        cudaStat = hipblasDaxpy(handle, matrix_dim,
                           &alpha,
                           (double*)(Q+i*matrix_dim), 1,
                           q, 1);
        cublasCheck(cudaStat,"hipblasDaxpy");

    }
    cudaStat = hipblasDnrm2( handle, matrix_dim,
                            q, 1, q_norm);
    cublasCheck(cudaStat,"hipblasDnrm2");

    element_append_vector<<<1,1,0,stream_id>>> (h, k, *q_norm);


    //seems there is a copy from device memory to host memory and is blocking
    double q_norm_reciprocal=1/(*q_norm);
    //printf("q_norm_reciprocal:%.3f\n",q_norm_reciprocal);

    // q = q / h(k + 1);
    cudaStat=hipblasSetStream(handle, stream_id);
    cublasCheck(cudaStat,"hipblasSetStream");

    cudaStat = hipblasDscal(handle, matrix_dim,
                            &q_norm_reciprocal,
                            q, 1);
    cublasCheck(cudaStat,"hipblasDnrm2");
    

    hipMemcpy((double*)(Q+(k+1)*matrix_dim),q,sizeof(double) *matrix_dim,hipMemcpyDefault);
    hipMemcpy((double*)(H+(k+1)*matrix_dim),h,sizeof(double) *matrix_dim,hipMemcpyDefault);
    
   
    hipFree(q_norm);
    hipFree(q);
    hipFree(h);

}

